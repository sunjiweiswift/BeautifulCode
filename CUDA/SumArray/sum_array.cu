
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void sum(float* input, float* result, int inputSize) {
    int index = threadIdx.x;
    __shared__ float sharedData[1024];
    sharedData[index] = input[index];
    for (int i = inputSize / 2; i > 0; i /= 2) {
        if (index < i) {
            sharedData[index] = sharedData[index] + sharedData[index + i];
        }
        __syncthreads();
    }
    if (index == 0) {
        *result = sharedData[0];
    }
}

int main(void) {
    int inputSize = 1024;
    float* inputCPU = new float[inputSize];
    for (int i = 0; i < inputSize; i++) {
        inputCPU[i] = i * (i + 1);
    }
    float* inputGPU;
    float* outputGPU;
    hipMalloc(&inputGPU, inputSize * sizeof(float));
    hipMalloc(&outputGPU, sizeof(float));
    hipMemcpy(inputGPU, inputCPU, inputSize * sizeof(float), hipMemcpyHostToDevice);
    sum<<<1, inputSize>>>(inputGPU, outputGPU, inputSize);
    float outputCPU;
    hipMemcpy(&outputCPU, outputGPU, sizeof(float), hipMemcpyDeviceToHost);
    printf("result: %f \n", outputCPU);
    return 0;
}
