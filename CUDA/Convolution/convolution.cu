
#include <hip/hip_runtime.h>
#include <stdio.h>

using namespace std;

__global__ void convolutionCUDA(float* img, float* kernel, float* result, int width, int height,
                                int kernelSize) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < width * height) {
        int row = index / width;
        int col = index % width;
        for (int i = 0; i < kernelSize; i++) {
            for (int j = 0; j < kernelSize; j++) {
                int curRow = row - kernelSize / 2 + i;
                int curCol = col - kernelSize / 2 + j;
                if (curRow >= 0 && curRow < height && curCol >= 0 && curCol < width) {
                    result[index] += img[curRow * width + curCol] * kernel[i * kernelSize + j];
                }
            }
        }
    }
}

int getThreadNum() {
    hipDeviceProp_t prop; //cudaDeviceProp的一个对象
    int count = 0;       //GPU的个数
    hipGetDeviceCount(&count);
    printf("gpu 的个数：%d \n", count);

    hipGetDeviceProperties(&prop, 0); //第二参数为那个gpu
    printf("最大线程数：%d \n", prop.maxThreadsPerBlock);
    printf("最大网格类型：%d %d %d \n", prop.maxGridSize[0], prop.maxGridSize[1],
           prop.maxGridSize[2]);
    return prop.maxThreadsPerBlock;
}

int main(void) {
    getThreadNum();
    int width = 1920;
    int height = 1080;
    float* imgCPU = new float[width * height];
    for (int row = 0; row < height; row++) {
        for (int col = 0; col < width; col++) {
            imgCPU[col + row * width] = (col + row) % 256;
        }
    }

    int kernelSize = 3;
    float* kernelCPU = new float[width * height];
    for (int i = 0; i < kernelSize * kernelSize; i++) {
        kernelCPU[i] = i % kernelSize - 1;
    }
    float* resultCPU = new float[width * height];
    // Print
    printf("Img data \n");
    for (int row = 0; row < 10; row++) {
        for (int col = 0; col < 10; col++) {
            printf("%2.0f ", imgCPU[col + row * width]);
        }
        printf("\n");
    }
    printf("Kernel data \n");
    for (int row = 0; row < kernelSize; row++) {
        for (int col = 0; col < kernelSize; col++) {
            printf("%2.0f ", kernelCPU[col + row * kernelSize]);
        }
        printf("\n");
    }
    // Malloc On GPU
    float* imgGPU;
    float* kernelGPU;
    float* resultGPU;
    hipMalloc(&imgGPU, width * height * sizeof(float));
    hipMalloc(&kernelGPU, kernelSize * kernelSize * sizeof(float));
    hipMalloc(&resultGPU, width * height * sizeof(float));
    // Copy CPU To CPU
    hipMemcpy(imgGPU, imgCPU, width * height * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(kernelGPU, kernelCPU, kernelSize * kernelSize * sizeof(float),
               hipMemcpyHostToDevice);
    int threadNum = 1024;
    int blockNum = (width * height + threadNum - 1) / threadNum;
    printf("threadNum:%d, blockNum:%d", threadNum, blockNum);

    convolutionCUDA<<<blockNum, threadNum>>>(imgGPU, kernelGPU, resultGPU, width, height,
                                             kernelSize);
    // Copy GPU to CPU
    hipMemcpy(resultCPU, resultGPU, width * height * sizeof(float), hipMemcpyDeviceToHost);
    // GPU cudaFree
    hipFree(imgGPU);
	hipFree(kernelGPU);
	hipFree(resultGPU);
    // Print
    printf("result data \n");
    for (int row = 0; row < 10; row++) {
        for (int col = 0; col < 10; col++) {
            printf("%2.0f ", resultCPU[col + row * width]);
        }
        printf("\n");
    }
    delete[] imgCPU;
    delete[] kernelCPU;
    delete[] resultCPU;
    return 0;
}
