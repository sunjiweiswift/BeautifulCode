#include "hip/hip_runtime.h"
#include <stdio.h>
#include <vector>

using namespace std;

__global__ void getMidNum(int* input, int size, int k, double* mid) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index <= size - k) {
        int* tmp = (int*)malloc(k * sizeof(int));
        memcpy(tmp, input + index, k * sizeof(int));

        // sort
        for (int i = 1, j; i < k; i++) {
            int insert = tmp[i];
            for (j = i - 1; j >= 0 && tmp[j] > insert; j--) {
                tmp[j + 1] = tmp[j];
            }
            tmp[j + 1] = insert;
        }

        // Get mid
        if (k % 2 == 1) {
            mid[index] = tmp[k / 2];
        } else {
            mid[index] = tmp[k / 2] / 2.0  + tmp[k - 1] / 2.0;
        }
        free tmp;
    }
}

int main(void) {
    // SetInput
    vector<int> cpuInputVec {1,3,-1,-3,5,3,6,7};
    int k = 3;
    // Generate variable
    int inputSize = cpuInputVec.size();
    int outputSize = inputSize - k + 1;
    int* cpuInput = cpuInputVec.data();

    double* cpuOutput = (double*)malloc(outputSize * sizeof(double));

    int* input;
    double* output;
    hipMalloc(&input, inputSize * sizeof(int));
    hipMalloc(&output, outputSize * sizeof(double));
    hipMemcpy(input, cpuInput, inputSize * sizeof(int), hipMemcpyHostToDevice);
    int threadPerBlock = 1;
    int numBlock = (inputSize - 1) / threadPerBlock + 1;
    // Run On GPU
    getMidNum<<<numBlock, threadPerBlock>>>(input, inputSize, k, output);
    hipMemcpy(cpuOutput, output, outputSize * sizeof(double), hipMemcpyDeviceToHost);
    for (int i = 0; i < outputSize; i++) {
        printf("%f ", cpuOutput[i]);
    }
    printf("\n");
    hipFree(input);
    hipFree(output);
    free(cpuOutput);
    hipDeviceReset();
    return 0;
}
