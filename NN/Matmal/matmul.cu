#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void matMulCUDA(float* A, float* B, float* C, int aHeight, int aWidth, int bWidth) {

    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    float sum = 0.0;
    if (row < aHeight && col < bWidth ) {
        for (int i = 0; i < aWidth; i++) {
            sum += A[row * aWidth + i] * B[i * bWidth + col];
        }
        C[row * bWidth + col] = sum;
    }
}

int main(void) {
    int aHeight = 2;
    int aWidth = 3;
    int bWidth = 3;
    float cpuA[aHeight][aWidth] = {{1, 2, 3}, {4, 5, 6}};
    float cpuB[aWidth][bWidth] = {{7, 8, 9}, {10, 11, 12}, {13, 14, 15}};
    // malloc GPU Memory
    float* A;
    float* B;
    float* C;
    hipMalloc(&A, aHeight * aWidth * sizeof(float));
    hipMalloc(&B, aWidth * bWidth * sizeof(float));
    hipMalloc(&C, aHeight * bWidth * sizeof(float));
    // Copy CPU to GPU
    hipMemcpy(A, cpuA, aHeight * aWidth * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(B, cpuB, aWidth * bWidth * sizeof(float), hipMemcpyHostToDevice);
    // Run On GPU
    dim3 blockSize(aHeight, bWidth);
    dim3 gridSize((aHeight + blockSize.x - 1) / blockSize.x,
                  (bWidth + blockSize.y - 1) / blockSize.y);
    matMulCUDA<<<gridSize, blockSize>>>(A, B, C, aHeight, aWidth, bWidth);
    // Copy GPU to CPU
    float* cpuC = (float*)malloc(aHeight * bWidth * sizeof(float));
    hipMemcpy(cpuC, C, aHeight * bWidth * sizeof(float), hipMemcpyDeviceToHost);
    for (int i = 0; i < aHeight; i++) {
        for (int j = 0; j < bWidth; j++) {
            printf("%f ", *(cpuC + i * bWidth + j));
        }
        printf("\n");
    }
    hipFree(A);
    hipFree(B);
    hipFree(C);
    free(cpuC);
    return 0;
}
