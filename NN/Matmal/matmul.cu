#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void matMulCUDA(float* A, float* B, float* C, int M, int K, int N) {

    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    float sum = 0.0;
    if (row < M && col < N ) {
        for (int i = 0; i < K; i++) {
            sum += A[row * K + i] * B[i * N + col];
        }
        C[row * N + col] = sum;
    }
}

int main(void) {
    int M = 2;
    int K = 3;
    int N = 3;
    float cpuA[M][K] = {{1, 2, 3}, {4, 5, 6}};
    float cpuB[K][N] = {{7, 8, 9}, {10, 11, 12}, {13, 14, 15}};
    // malloc GPU Memory
    float* A;
    float* B;
    float* C;
    hipMalloc(&A, M * K * sizeof(float));
    hipMalloc(&B, K * N * sizeof(float));
    hipMalloc(&C, M * N * sizeof(float));
    // Copy CPU to GPU
    hipMemcpy(A, cpuA, M * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(B, cpuB, K * N * sizeof(float), hipMemcpyHostToDevice);
    // Run On GPU
    dim3 blockSize(M, N);
    dim3 gridSize((M + blockSize.x - 1) / blockSize.x,
                  (N + blockSize.y - 1) / blockSize.y);
    matMulCUDA<<<gridSize, blockSize>>>(A, B, C, M, K, N);
    // Copy GPU to CPU
    float* cpuC = (float*)malloc(M * N * sizeof(float));
    hipMemcpy(cpuC, C, M * N * sizeof(float), hipMemcpyDeviceToHost);
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            printf("%f ", *(cpuC + i * N + j));
        }
        printf("\n");
    }
    hipFree(A);
    hipFree(B);
    hipFree(C);
    free(cpuC);
    return 0;
}
